#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.h"
#include "pmcl3d_cons.h"
#include "cuda_to_hip.h"

__constant__ float d_c1;
__constant__ float d_c2;
__constant__ float d_dth;
__constant__ float d_dt1;
__constant__ float d_dh1;
__constant__ float d_DT;
__constant__ float d_DH;
__constant__ int   d_nxt;
__constant__ int   d_nyt;
__constant__ int   d_nzt;
__constant__ int   d_slice_1;
__constant__ int   d_slice_2;
__constant__ int   d_yline_1;
__constant__ int   d_yline_2;

texture<float, 1, hipReadModeElementType> p_vx1;
texture<float, 1, hipReadModeElementType> p_vx2;

extern "C"
void SetDeviceConstValue(float DH, float DT, int nxt, int nyt, int nzt)
{
    float h_c1, h_c2, h_dth, h_dt1, h_dh1;
    int   slice_1,  slice_2,  yline_1,  yline_2;
    h_c1  = 9.0/8.0;
    h_c2  = -1.0/24.0;
    h_dth = DT/DH;
    h_dt1 = 1.0/DT;
    h_dh1 = 1.0/DH;
    slice_1  = (nyt+4+8*loop)*(nzt+2*align);
    slice_2  = (nyt+4+8*loop)*(nzt+2*align)*2;
    yline_1  = nzt+2*align;
    yline_2  = (nzt+2*align)*2;
  
    hipMemcpyToSymbol(HIP_SYMBOL(d_c1),      &h_c1,    sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_c2),      &h_c2,    sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dth),     &h_dth,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dt1),     &h_dt1,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dh1),     &h_dh1,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_DT),      &DT,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_DH),      &DH,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nxt),     &nxt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nyt),     &nyt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nzt),     &nzt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_slice_1), &slice_1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_slice_2), &slice_2, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_yline_1), &yline_1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_yline_2), &yline_2, sizeof(int));
    return;
}

extern "C"
void BindArrayToTexture(float* vx1, float* vx2, int memsize)
{
   hipBindTexture(0, p_vx1,  vx1,  memsize);
   hipBindTexture(0, p_vx2,  vx2,  memsize);
   hipDeviceSynchronize ();
   return;
}

extern "C"
void UnBindArrayFromTexture()
{
   hipUnbindTexture(p_vx1);
   hipUnbindTexture(p_vx2);
   return;
}

extern "C"
void dvelcx_H(float* u1,    float* v1,    float* w1,    float* xx,  float* yy, float* zz, float* xy,      float* xz, float* yz,
             float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, int nyt,   int nzt,   hipStream_t St, int s_i,   int e_i, int rank)
{
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nyt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvelcx), hipFuncCachePreferL1);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dvelcx, before kernel: %s\n",rank,hipGetErrorString(err));
    dvelcx<<<grid, block, 0, St>>>(u1, v1, w1, xx, yy, zz, xy, xz, yz, dcrjx, dcrjy, dcrjz, d_1, s_i, e_i);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dvelcx: %s\n",rank,hipGetErrorString(err));
    return;
}

extern "C"
void dvelcy_H(float* u1,       float* v1,    float* w1,    float* xx,  float* yy, float* zz, float* xy,   float* xz,   float* yz,
              float* dcrjx,    float* dcrjy, float* dcrjz, float* d_1, int nxt,   int nzt,   float* s_u1, float* s_v1, float* s_w1,  
              hipStream_t St, int s_j,      int e_j,      int rank,   int rank_me)
{
    if(rank==-1) return;
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nxt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvelcy), hipFuncCachePreferL1);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dvelcy, before kernel: %s\n",rank_me,hipGetErrorString(err));
    dvelcy<<<grid, block, 0, St>>>(u1, v1, w1, xx, yy, zz, xy, xz, yz, dcrjx, dcrjy, dcrjz, d_1, s_u1, s_v1, s_w1, s_j, e_j);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dvelcy: %s\n",rank_me,hipGetErrorString(err));
    return;
}

extern "C"
void update_bound_y_H(float* u1,   float* v1, float* w1, float* f_u1,      float* f_v1,      float* f_w1,  float* b_u1, float* b_v1, 
                      float* b_w1, int nxt,   int nzt,   hipStream_t St1, hipStream_t St2, int rank_f,  int rank_b, int rank)
{
     if(rank_f==-1 && rank_b==-1) return;
     dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
     dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nxt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
     hipFuncSetCacheConfig(reinterpret_cast<const void*>(update_boundary_y), hipFuncCachePreferL1);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d update_boundary, before kernel: %s\n",rank,hipGetErrorString(err));
     update_boundary_y<<<grid, block, 0, St1>>>(u1, v1, w1, f_u1, f_v1, f_w1, rank_f, Front);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d update_boundary, intermediate: %s\n",rank,hipGetErrorString(err));
     update_boundary_y<<<grid, block, 0, St2>>>(u1, v1, w1, b_u1, b_v1, b_w1, rank_b, Back);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! update_boundary: %s\n",hipGetErrorString(err));
     return;
}

extern "C"
void dstrqc_H(float* xx,       float* yy,     float* zz,    float* xy,    float* xz, float* yz,
              float* r1,       float* r2,     float* r3,    float* r4,    float* r5, float* r6,
              float* u1,       float* v1,     float* w1,    float* lam,   float* mu, float* qp,
              float* qs,       float* dcrjx,  float* dcrjy, float* dcrjz, int nyt,   int nzt, 
              hipStream_t St, float* lam_mu, int NX,       int rankx,    int ranky, int  s_i,  
              int e_i,         int s_j,       int e_j,      int rank)
{
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (e_j-s_j+1+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dstrqc), hipFuncCachePreferL1);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dstrqc, before kernel: %s\n",rank,hipGetErrorString(err));
    dstrqc<<<grid, block, 0, St>>>(xx,    yy,    zz,  xy,  xz, yz, r1, r2,    r3,    r4,    r5,     r6, 
                                   u1,    v1,    w1,  lam, mu, qp, qs, dcrjx, dcrjy, dcrjz, lam_mu, NX, 
                                   rankx, ranky, s_i, e_i, s_j);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! rank=%d dstrqc: %s\n",rank,hipGetErrorString(err));
    return;
}

extern "C"
void addsrc_H(int i,      int READ_STEP, int dim,    int* psrc,  int npsrc,  hipStream_t St,
              int igreen, int nzt, 
              float* d1,     float* u1,  float* v1,  float* w1,
              float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
              float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz)
{
    dim3 grid, block;
    if(npsrc < 256)
    {
       block.x = npsrc;
       grid.x = 1;
    }
    else
    {
       block.x = 256;
       grid.x  = int((npsrc+255)/256);
    }
/*
    int nyt=420, nxt=310;
    int i_=2+4*loop,j=2+4*loop,k=align;
    int pos = i_*(nyt+4+8*loop)*(nzt+2*align)+j*(nzt+2*align)+k;
printf("xx,yy,zz,xy,xz,yz=%f,%f,%f,%f,%f,%f\n",
      xx[pos],yy[pos],zz[pos],xy[pos],xz[pos],yz[pos]);
*/
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! addsrc_H, before addsrc_cu: %s\n",hipGetErrorString(err));
    addsrc_cu<<<grid, block, 0, St>>>(i,   READ_STEP, dim, npsrc, psrc, igreen, nzt, 
                                      d1,  u1,        v1,  w1,
                                      axx, ayy,       azz, axz,  ayz,    axy,
                                      xx,  yy,        zz,  xy,   yz,     xz);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! addsrc_H: %s\n",hipGetErrorString(err));
    return;
}

extern "C"
void addkinsrc_H(int i,   int dim,    int* psrc,  int npsrc,  hipStream_t St, float* mu,
              float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
              float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz,
              float* mom)
{
    dim3 grid, block;
    if(npsrc < 256)
    {
       block.x = npsrc;
       grid.x = 1;
    }
    else
    {
       block.x = 256;
       grid.x  = int((npsrc+255)/256);
    }
    hipError_t cerr;
    cerr=hipGetLastError();
    if(cerr!=hipSuccess) printf("CUDA ERROR: addkinsrc before kernel: %s\n",hipGetErrorString(cerr));
    //cudaPrintfInit();
	printf("grid.x=%d, block.x=%d\n", grid.x, block.x);
    addkinsrc_cu<<<grid, block, 0, St>>>(i, dim, psrc, npsrc, mu, axx, ayy, azz, axz, ayz, axy,
                                      xx, yy, zz,  xy,   yz,  xz, mom);
    cerr=hipGetLastError();
    /*cudaPrintfDisplay(stdout, 1);
    cudaPrintfEnd();*/
    if(cerr!=hipSuccess) printf("CUDA ERROR: addkinsrc after kernel: %s\n",hipGetErrorString(cerr));
    return;
}

__device__ void compmt(float str, float dip, float rake,
        float *xx, float *yy, float *zz, float *xz, float *yz, float *xy ){

      //angles must be provided in rads

      *yy= -(sinf(dip)*cosf(rake)*sinf(2.*str)+
           sinf(2.*dip)*sinf(rake)*sinf(str)*sinf(str));

      *xy= sinf(dip)*cosf(rake)*cosf(2.*str)+
           0.5*(sinf(2.*dip)*sinf(rake)*sinf(2.*str));

      *yz= (cosf(dip)*cosf(rake)*cosf(str)+
           cosf(2.*dip)*sinf(rake)*sinf(str));

      *xx= sinf(dip)*cosf(rake)*sinf(2.*str)-
           sinf(2.*dip)*sinf(rake)*cosf(str)*cosf(str);

      *xz= (cosf(dip)*cosf(rake)*sinf(str)-
           cosf(2.*dip)*sinf(rake)*cosf(str));

      *zz= sinf(2.*dip)*sinf(rake);
}

__device__ float brune(float freq, float time){
   register float stf, omega;
   omega=freq * M_PI * 2.;
   if (time > 0.)
      stf = powf(omega, 2.) * time * expf(-omega*time);
   else
      stf = 0.;
   return(stf);
}

/* Liu et al. (2006) source time function.  tau = risetime */
__device__ float liu(float tau, float time){
   register float tau1, tau2, CN, stf;

   tau1 = 0.13 * tau;
   tau2 = tau-tau1;

   CN=M_PI / (1.4*M_PI*tau1 + 1.2*tau1 + 0.3*M_PI*tau2);

   if (time < tau1)
      stf = CN*(0.7 - 0.7*cosf(M_PI*time/tau1) + 0.6*sinf(0.5*M_PI*time/tau1));
   else if (time < 2*tau1)
      stf = CN*(1.0 - 0.7*cosf(M_PI*time/tau1) + 0.3*cosf(M_PI*(time-tau1)/tau2));
   else if (time < tau)
      stf = CN*(0.3 + 0.3*cosf(M_PI*(time-tau1) / tau2));
   else
      stf = 0.;

   return(stf);
}

/* GP 2010, revised based on Liu et al. (2006) source time function.  tau = risetime */
__device__ float gp10(float tau, float time){
   register float tau1, tau2, CN, stf;

   tau1 = 0.13 * tau;
   tau2 = tau-tau1;

   CN=M_PI / (1.5 * M_PI*tau1 + 1.2*tau1 + 0.2 * M_PI * tau2);
   if (time < 0.)
      stf = 0.;
   else if (time < tau1)
      stf = CN*(0.7 - 0.7*cosf(M_PI*time/tau1) + 0.6*sinf(0.5*M_PI*time/tau1));
   else if (time < 2*tau1)
      stf = CN*(1.0 - 0.8*cosf(M_PI*time/tau1) + 0.2*cosf(M_PI*(time-tau1)/tau2));
   else if (time < tau)
      stf = CN*(0.2 + 0.2*cosf(M_PI*(time-tau1) / tau2));
   else
      stf = 0.;

   return(stf);
}



__global__ void addkinsrc_cu(int i, int dim,    int* psrc,  int npsrc, float* mu,
                          float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
                          float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz,
                          float* mom)
{

        register float vtst;
        register int idx, idy, idz, j, pos;

        register float atime, freq, stf;
        register float axxt, ayyt, azzt, axzt, ayzt, axyt;
        register int stf_type;
        register float slip, ruptime, risetime, strike, dip, rake, area;

        register int READ_STEP = 2;
        register double *stff[MAXFILT];
        int n;

        j = blockIdx.x*blockDim.x+threadIdx.x;
        if(j >= npsrc) return;

        // For a kinematic source, the moment-rate is computed at run-time from given subfault parameters,
        // which are stored inside the arrays axx...axz 
        stf_type = (int) axx[j*READ_STEP]; // type of source time function.  1=Brune
        slip = ayy[j*READ_STEP];     // total slip
        ruptime = azz[j*READ_STEP];  // rupture time
        risetime = axz[j*READ_STEP]; // rise time

        atime = i*d_DT;

        if (atime > ruptime) {
       area = ayz[j*READ_STEP];   // subfault area

       strike = axx[j*READ_STEP+1] / 180. * M_PI;   // strike angle (given in degrees)
       dip = ayy[j*READ_STEP+1] / 180. * M_PI;      // dip angle
       rake = azz[j*READ_STEP+1] / 180. * M_PI;     // rake

           compmt(strike, dip, rake, &axxt, &ayyt, &azzt, &axzt, &ayzt, &axyt);

       if (stf_type == 1.0f){
          freq = 1./risetime;
          stf = brune(freq, atime - ruptime);
       }
       else if (stf_type == 2.0f)
          stf = liu(risetime, atime - ruptime);
       else if (stf_type == 3.0f)
        stf = gp10(risetime, atime - ruptime);
     else
          stf = 0.;

       vtst = (float)d_DT/(d_DH*d_DH*d_DH);


        idx = psrc[j*dim]   + 1 + 4*loop;
        idy = psrc[j*dim+1] + 1 + 4*loop;
        idz = psrc[j*dim+2] + align - 1;
        pos = idx*d_slice_1 + idy*d_yline_1 + idz;

       /*idx = psrc[j*dim]   + 1 + ngsl;
       idy = psrc[j*dim+1] + 1 + ngsl;
       idz = psrc[j*dim+2] + align - 1;
       pos = idx*d_slice_1 + idy*d_yline_1 + idz;*/

           stf *= slip*area/mu[pos];
           mom[j] += stf * d_DT;

           //cuPrintf("stf: %d %e %e %e %e\n", j, atime, stf, slip, area);
           //cuPrintf("mom: %d %e\n", j, mom[j]);

           stf *= vtst;

           /*if (j == 0)
          cuPrintf("addkinsrc_cu: (%d,%d,%d) (%e, %e,%e,%e,%e,%e,%e)\n", idx, idy, idz, 
             stf, axxt, ayyt, azzt, axzt, ayzt, axyt);
          cuPrintf("addkinsrc_cu: (%d,%d,%d) (%e, %e, %e m^2, %f m)\n", idx, idy, idz, 
             stf, 1./mu[pos], area, slip);*/

       xx[pos] = xx[pos] - stf*axxt;
       yy[pos] = yy[pos] - stf*ayyt;
       zz[pos] = zz[pos] - stf*azzt;
       xz[pos] = xz[pos] - stf*axzt;
       yz[pos] = yz[pos] - stf*ayzt;
       xy[pos] = xy[pos] - stf*axyt;

        }

        return;
}


extern "C"
void ComputeSGT(float* xx,   float* yy,        float* zz,    float* xy,    float* xz,     float* yz,
//                float* sgt1, float* sgt2,      float* sgt3,  float* sgt4,  float* sgt5,   float* sgt6,
                float* sg1,  float* sg2,       float* mu,    int sgt_numsta, int* sgt_sta,float* sgtBuf,     
                hipStream_t St,  int nzt,     int SGT_BLOCK_SIZE, int SGT_NUMBLOCKS, float* qs, float* d1)

{
    //dim3 block (BLOCK_SIZE_Z, 1, 1);
    //dim3 grid ((sgt_numsta+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (e_j-s_j+1+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    dim3 block (SGT_BLOCK_SIZE, 1, 1);
    dim3 grid (SGT_NUMBLOCKS, 1, 1);
/*
    int nyt=420, nxt=310;
    int i=2+4*loop,j=2+4*loop,k=align;
    int pos = i*(nyt+4+8*loop)*(nzt+2*align)+j*(nzt+2*align)+k;
printf("pos=%d\n",pos);
printf("xx,yy,zz,xy,xz,yz=%f,%f,%f,%f,%f,%f\n",
      xx[pos],yy[pos],zz[pos],xy[pos],xz[pos],yz[pos]);
printf("sg1,sg2,mu=%f,%f,%f\n",
      sg1[pos],sg2[pos],mu[pos]);
printf("sgt_n,sgt_st,sgtBuf=%d,(%d,%d,%d),%f\n",
      sgt_numsta, sgt_sta[0],sgt_sta[1],sgt_sta[2], sgtBuf[0]);
printf("BS,NB=%d,%d\n",
      SGT_BLOCK_SIZE, SGT_NUMBLOCKS);
*/
    size_t cmemfree, cmemtotal;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ComputeSGT_cu), hipFuncCachePreferL1);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! ComputeSGT before kernel: %s\n",hipGetErrorString(err));
    //        hipMemGetInfo(&cmemfree, &cmemtotal);
    //        printf("CUDA memory, inside ComputeSGT numsta=%d, free=%ld, total=%ld\n",sgt_numsta,cmemfree,cmemtotal);
    ComputeSGT_cu<<<grid, block, 0, St>>>(xx,    yy,    zz,   xy,   xz,   yz, 
//                                          sgt1,  sgt2,  sgt3, sgt4, sgt5, sgt6,
                                          sg1,   sg2,   mu,   sgt_numsta, sgt_sta, sgtBuf,
                                          SGT_BLOCK_SIZE, SGT_NUMBLOCKS, qs, d1);
    //        hipMemGetInfo(&cmemfree, &cmemtotal);
    //        printf("CUDA memory, inside ComputeSGT, numsta=%d, free=%ld, total=%ld\n",sgt_numsta,cmemfree,cmemtotal);
    err = hipGetLastError();
    if(err != hipSuccess) printf("CUDA KERNEL ERROR! ComputeSGT: %s\n",hipGetErrorString(err));
    return;
}


__global__ void dvelcx(float* u1,    float* v1,    float* w1,    float* xx, float* yy, float* zz, float* xy, float* xz, float* yz, 
                      float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, int s_i,   int e_i)
{
    register int   i, j, k, pos,     pos_im1, pos_im2;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_jm2, pos_jm1, pos_jp1, pos_jp2;
    register int   pos_ip1, pos_jk1, pos_ik1, pos_ijk;
    register float f_xx,    xx_im1,  xx_ip1,  xx_im2;
    register float f_xy,    xy_ip1,  xy_ip2,  xy_im1;
    register float f_xz,    xz_ip1,  xz_ip2,  xz_im1;
    register float f_d1,    f_d2,    f_d3,    f_dcrj, f_dcrjy, f_dcrjz, f_yz;

    k    = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+align;
    j    = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;
    i    = e_i;
    pos  = i*d_slice_1+j*d_yline_1+k;

    f_xx    = xx[pos+d_slice_1];
    xx_im1  = xx[pos];
    xx_im2  = xx[pos-d_slice_1]; 
    xy_ip1  = xy[pos+d_slice_2];
    f_xy    = xy[pos+d_slice_1];
    xy_im1  = xy[pos];
    xz_ip1  = xz[pos+d_slice_2];
    f_xz    = xz[pos+d_slice_1];
    xz_im1  = xz[pos];
    f_dcrjz = dcrjz[k];
    f_dcrjy = dcrjy[j]; 
    for(i=e_i;i>=s_i;i--)   
    {
        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_jp1  = pos+d_yline_1;
        pos_jp2  = pos+d_yline_2;
        pos_im1  = pos-d_slice_1;
        pos_im2  = pos-d_slice_2;
        pos_ip1  = pos+d_slice_1;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;

        xx_ip1   = f_xx;
        f_xx     = xx_im1;
        xx_im1   = xx_im2;
        xx_im2   = xx[pos_im2];
        xy_ip2   = xy_ip1;
        xy_ip1   = f_xy;
        f_xy     = xy_im1;
        xy_im1   = xy[pos_im1];
        xz_ip2   = xz_ip1;
        xz_ip1   = f_xz;
        f_xz     = xz_im1;
        xz_im1   = xz[pos_im1];
        f_yz     = yz[pos];

        f_dcrj   = dcrjx[i]*f_dcrjy*f_dcrjz;
        f_d1     = 0.25*(d_1[pos] + d_1[pos_jm1] + d_1[pos_km1] + d_1[pos_jk1]);
        f_d2     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_km1] + d_1[pos_ik1]);
        f_d3     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_jm1] + d_1[pos_ijk]);

        f_d1     = d_dth/f_d1;
        f_d2     = d_dth/f_d2;
	f_d3     = d_dth/f_d3;

    	u1[pos]  = (u1[pos] + f_d1*( d_c1*(f_xx        - xx_im1)      + d_c2*(xx_ip1      - xx_im2) 
                                   + d_c1*(f_xy        - xy[pos_jm1]) + d_c2*(xy[pos_jp1] - xy[pos_jm2])
                                   + d_c1*(f_xz        - xz[pos_km1]) + d_c2*(xz[pos_kp1] - xz[pos_km2]) ))*f_dcrj; 
        v1[pos]  = (v1[pos] + f_d2*( d_c1*(xy_ip1      - f_xy)        + d_c2*(xy_ip2      - xy_im1)
                                   + d_c1*(yy[pos_jp1] - yy[pos])     + d_c2*(yy[pos_jp2] - yy[pos_jm1])
                                   + d_c1*(f_yz        - yz[pos_km1]) + d_c2*(yz[pos_kp1] - yz[pos_km2]) ))*f_dcrj;

        w1[pos]  = (w1[pos] + f_d3*( d_c1*(xz_ip1      - f_xz)        + d_c2*(xz_ip2      - xz_im1)
                                   + d_c1*(f_yz        - yz[pos_jm1]) + d_c2*(yz[pos_jp1] - yz[pos_jm2])
                                   + d_c1*(zz[pos_kp1] - zz[pos])     + d_c2*(zz[pos_kp2] - zz[pos_km1]) ))*f_dcrj;
        pos      = pos_im1;
    }

    return;
}


__global__ void dvelcy(float* u1,    float* v1,    float* w1,    float* xx,  float* yy,   float* zz,   float* xy, float* xz, float* yz,
                       float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, float* s_u1, float* s_v1, float* s_w1, int s_j,   int e_j)
{
    register int   i, j, k, pos,     j2,      pos2, pos_jm1, pos_jm2;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_im2, pos_im1, pos_ip1, pos_ip2;
    register int   pos_jk1, pos_ik1, pos_ijk;
    register float f_xy,    xy_jp1,  xy_jm1,  xy_jm2;
    register float f_yy,    yy_jp2,  yy_jp1,  yy_jm1;
    register float f_yz,    yz_jp1,  yz_jm1,  yz_jm2;
    register float f_d1,    f_d2,    f_d3,    f_dcrj, f_dcrjx, f_dcrjz, f_xz;

    k     = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+align;
    i     = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;
    j     = e_j;
    j2    = 4*loop-1;
    pos   = i*d_slice_1+j*d_yline_1+k;
    pos2  = i*4*loop*d_yline_1+j2*d_yline_1+k; 

    f_xy    = xy[pos+d_yline_1];
    xy_jm1  = xy[pos];
    xy_jm2  = xy[pos-d_yline_1];
    yy_jp1  = yy[pos+d_yline_2];
    f_yy    = yy[pos+d_yline_1];
    yy_jm1  = yy[pos];
    f_yz    = yz[pos+d_yline_1];
    yz_jm1  = yz[pos];
    yz_jm2  = yz[pos-d_yline_1];
    f_dcrjz = dcrjz[k];
    f_dcrjx = dcrjx[i];
    for(j=e_j; j>=s_j; j--)
    {
        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_im1  = pos-d_slice_1;
        pos_im2  = pos-d_slice_2;
        pos_ip1  = pos+d_slice_1;
        pos_ip2  = pos+d_slice_2;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;

        xy_jp1   = f_xy;
        f_xy     = xy_jm1;
        xy_jm1   = xy_jm2;
        xy_jm2   = xy[pos_jm2];
        yy_jp2   = yy_jp1;
        yy_jp1   = f_yy;
        f_yy     = yy_jm1;
        yy_jm1   = yy[pos_jm1];
        yz_jp1   = f_yz;
        f_yz     = yz_jm1;
        yz_jm1   = yz_jm2;
        yz_jm2   = yz[pos_jm2];
        f_xz     = xz[pos];

        f_dcrj   = f_dcrjx*dcrjy[j]*f_dcrjz;
        f_d1     = 0.25*(d_1[pos] + d_1[pos_jm1] + d_1[pos_km1] + d_1[pos_jk1]);
        f_d2     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_km1] + d_1[pos_ik1]);
        f_d3     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_jm1] + d_1[pos_ijk]);

        f_d1     = d_dth/f_d1;
        f_d2     = d_dth/f_d2;
        f_d3     = d_dth/f_d3;

        s_u1[pos2] = (u1[pos] + f_d1*( d_c1*(xx[pos]     - xx[pos_im1]) + d_c2*(xx[pos_ip1] - xx[pos_im2])
                                     + d_c1*(f_xy        - xy_jm1)      + d_c2*(xy_jp1      - xy_jm2)
                                     + d_c1*(f_xz        - xz[pos_km1]) + d_c2*(xz[pos_kp1] - xz[pos_km2]) ))*f_dcrj;
        s_v1[pos2] = (v1[pos] + f_d2*( d_c1*(xy[pos_ip1] - f_xy)        + d_c2*(xy[pos_ip2] - xy[pos_im1])
                                     + d_c1*(yy_jp1      - f_yy)        + d_c2*(yy_jp2      - yy_jm1)
                                     + d_c1*(f_yz        - yz[pos_km1]) + d_c2*(yz[pos_kp1] - yz[pos_km2]) ))*f_dcrj;
        s_w1[pos2] = (w1[pos] + f_d3*( d_c1*(xz[pos_ip1] - f_xz)        + d_c2*(xz[pos_ip2] - xz[pos_im1])
                                     + d_c1*(f_yz        - yz_jm1)      + d_c2*(yz_jp1      - yz_jm2)
                                     + d_c1*(zz[pos_kp1] - zz[pos])     + d_c2*(zz[pos_kp2] - zz[pos_km1]) ))*f_dcrj;

        pos        = pos_jm1;
        pos2       = pos2 - d_yline_1;
    }
    return;
}

__global__ void update_boundary_y(float* u1, float* v1, float* w1, float* s_u1, float* s_v1, float* s_w1, int rank, int flag)
{
    register int i, j, k, pos, posj;
    k     = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+align;
    i     = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;

    if(flag==Front && rank!=-1){
	j     = 2;
    	pos   = i*d_slice_1+j*d_yline_1+k;
        posj  = i*4*loop*d_yline_1+k;
	for(j=2;j<2+4*loop;j++){
		u1[pos] = s_u1[posj];
		v1[pos] = s_v1[posj];
		w1[pos] = s_w1[posj];
		pos	= pos  + d_yline_1;
  		posj	= posj + d_yline_1;	
	}
    }

    if(flag==Back && rank!=-1){
    	j     = d_nyt+4*loop+2;
    	pos   = i*d_slice_1+j*d_yline_1+k;
        posj  = i*4*loop*d_yline_1+k;
	for(j=d_nyt+4*loop+2;j<d_nyt+8*loop+2;j++){
	        u1[pos] = s_u1[posj];
                v1[pos] = s_v1[posj];
                w1[pos] = s_w1[posj];
                pos     = pos  + d_yline_1;
                posj    = posj + d_yline_1;
	}
    }
    return;
}

__global__ void ComputeSGT_cu(float* xx,   float* yy,    float* zz,    float* xy,    float* xz,     float* yz,
//                              float* sgt1, float* sgt2,  float* sgt3,  float* sgt4,  float* sgt5,   float* sgt6,
                              float* sg1,  float* sg2,   float* mu,    int sgt_numsta, int* sgt_sta,float* sgtBuf,
                              int SGT_BLOCK_SIZE,        int SGT_NUMBLOCKS, float* qs, float* d1)
{
    register int ind, i, j, k, pos;

    //if(SGT_NUMBLOCKS==1)
      //ind = threadIdx.x;
    //else
      ind = blockIdx.x*SGT_BLOCK_SIZE + threadIdx.x;
    if(sgt_numsta <= ind) return;
    i = sgt_sta[ind*3]   + 1 + 4*loop;
    j = sgt_sta[ind*3+1] + 1 + 4*loop;
    k = sgt_sta[ind*3+2] + align - 1;
    pos  = i*d_slice_1+j*d_yline_1+k;
    ind  = ind*6;
//    for(i=e_i;i>=s_i;i--)
//    {
/*
        (*sgtBuf)[ind] = sg1[pos]*xx[pos] + sg2[pos]*yy[pos] + sg2[pos]*zz[pos];
        ind++;
        (*sgtBuf)[ind] = sg2[pos]*xx[pos] + sg1[pos]*yy[pos] + sg2[pos]*zz[pos];
        ind++;
        (*sgtBuf)[ind] = sg2[pos]*xx[pos] + sg2[pos]*yy[pos] + sg1[pos]*zz[pos];
        ind++;
        (*sgtBuf)[ind] = mu[pos]*xy[pos];
        ind++;
        (*sgtBuf)[ind] = mu[pos]*xz[pos];
        ind++;
        (*sgtBuf)[ind] = mu[pos]*yz[pos];
*/
		//Recompute sg1, sg2, mu to use uncorrected values
		float FP = 1.0;
		float FL = 0.01;
		float FH = 25.0;
		float pi = 3.14159;
		float w0 = 2*pi*FP;
		float ww1 = 2*pi*FL;
		float w2 = 2*pi*FH;
		float taumax = 1.0/ww1;
		float taumin = 1.0/w2;
		float tmp1 = 2.0/pi*log(taumax/taumin);
		float tmp2 = 2.0/pi*log(w0*taumin);
		//Assumes Qp = 2Qs
		float orig_qs = tmp1/qs[pos]+tmp2;
		float orig_qp = 2*orig_qs;
		float vs_scalefac = 1 + log(w2/w0)/(pi*orig_qs);
		float vp_scalefac = 1 + log(w2/w0)/(pi*orig_qp);
		float orig_vs = sqrt(1.0/(mu[pos]*d1[pos]))/vs_scalefac;
		float lam = 0.5*(((-1.0*mu[pos])/(2.0*sg2[pos])-3.0)*mu[pos]);
		float orig_vp = sqrt(1.0/(d1[pos]*lam)+2*vs_scalefac*orig_vs*vs_scalefac*orig_vs)/vp_scalefac;
		float local_mu = mu[pos]*vs_scalefac*vs_scalefac;
		float local_lam = 1.0/(d1[pos]*(orig_vp*orig_vp-2.0*orig_vs*orig_vs));
		float local_sg1 = (1.0/local_lam+1.0/local_mu)/(1.0/local_mu*(3.0/local_lam+2.0/local_mu));
		float local_sg2 = -1.0/local_lam/(2.0/local_mu*(3.0/local_lam+2.0/local_mu));


        /*sgtBuf[ind] = sg1[pos]*xx[pos] + sg2[pos]*yy[pos] + sg2[pos]*zz[pos];
        ind++;
        sgtBuf[ind] = sg2[pos]*xx[pos] + sg1[pos]*yy[pos] + sg2[pos]*zz[pos];
        ind++;
        sgtBuf[ind] = sg2[pos]*xx[pos] + sg2[pos]*yy[pos] + sg1[pos]*zz[pos];
        ind++;
        sgtBuf[ind] = mu[pos]*xy[pos];
        ind++;
        sgtBuf[ind] = mu[pos]*xz[pos];
        ind++;
        sgtBuf[ind] = mu[pos]*yz[pos];*/

		sgtBuf[ind] = local_sg1*xx[pos] + local_sg2*yy[pos] + local_sg2*zz[pos];
        ind++;
        sgtBuf[ind] = local_sg2*xx[pos] + local_sg1*yy[pos] + local_sg2*zz[pos];
        ind++;
        sgtBuf[ind] = local_sg2*xx[pos] + local_sg2*yy[pos] + local_sg1*zz[pos];
        ind++;
        sgtBuf[ind] = local_mu*xy[pos];
        ind++;
        sgtBuf[ind] = local_mu*xz[pos];
        ind++;
        sgtBuf[ind] = local_mu*yz[pos];
		
		//To output strains
        /*sgtBuf[ind] = xx[pos];
        ind++;
        sgtBuf[ind] = yy[pos];
        ind++;
        sgtBuf[ind] = zz[pos];
        ind++;
        sgtBuf[ind] = xy[pos];
        ind++;
        sgtBuf[ind] = xz[pos];
        ind++;
        sgtBuf[ind] = yz[pos];*/
//        pos       = pos - d_slice_1;
//    }
    return;
}


__global__ void dstrqc(float* xx, float* yy,    float* zz,    float* xy,    float* xz,     float* yz,
                       float* r1, float* r2,    float* r3,    float* r4,    float* r5,     float* r6,
                       float* u1, float* v1,    float* w1,    float* lam,   float* mu,     float* qp,
                       float* qs, float* dcrjx, float* dcrjy, float* dcrjz, float* lam_mu, int NX,    
                       int rankx, int ranky,    int s_i,      int e_i,      int s_j)
{
    register int   i,  j,  k,  g_i;
    register int   pos,     pos_ip1, pos_im2, pos_im1;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_jm2, pos_jm1, pos_jp1, pos_jp2;
    register int   pos_ik1, pos_jk1, pos_ijk, pos_ijk1;
    register float vs1, vs2, vs3, a1, tmp, vx1;
    register float xl,  xm,  xmu1, xmu2, xmu3;
    register float qpa, h,   h1,   h2,   h3;
    register float f_vx1, f_vx2,  f_dcrj, f_r,  f_dcrjy, f_dcrjz;
    register float f_u1, u1_ip1, u1_ip2, u1_im1;
    register float f_v1, v1_im1, v1_ip1, v1_im2;
    register float f_w1, w1_im1, w1_im2, w1_ip1;
    
    k    = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+align;
    j    = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+s_j;
    i    = e_i;
    pos  = i*d_slice_1+j*d_yline_1+k;

    u1_ip1 = u1[pos+d_slice_2];
    f_u1   = u1[pos+d_slice_1];
    u1_im1 = u1[pos];    
    f_v1   = v1[pos+d_slice_1];
    v1_im1 = v1[pos];
    v1_im2 = v1[pos-d_slice_1];
    f_w1   = w1[pos+d_slice_1];
    w1_im1 = w1[pos];
    w1_im2 = w1[pos-d_slice_1];
    f_dcrjz = dcrjz[k];
    f_dcrjy = dcrjy[j];
    for(i=e_i;i>=s_i;i--)
    {
        f_vx1    = tex1Dfetch(p_vx1, pos);
        f_vx2    = tex1Dfetch(p_vx2, pos);
        f_dcrj   = dcrjx[i]*f_dcrjy*f_dcrjz;

        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_jp1  = pos+d_yline_1;
        pos_jp2  = pos+d_yline_2;
        pos_im2  = pos-d_slice_2;
        pos_im1  = pos-d_slice_1;
        pos_ip1  = pos+d_slice_1;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;
        pos_ijk1 = pos+d_slice_1-d_yline_1-1;

        xl       = 8.0/(  lam[pos]      + lam[pos_ip1] + lam[pos_jm1] + lam[pos_ijk]
                        + lam[pos_km1]  + lam[pos_ik1] + lam[pos_jk1] + lam[pos_ijk1] );
        xm       = 16.0/( mu[pos]       + mu[pos_ip1]  + mu[pos_jm1]  + mu[pos_ijk]
                        + mu[pos_km1]   + mu[pos_ik1]  + mu[pos_jk1]  + mu[pos_ijk1] );
        xmu1     = 2.0/(  mu[pos]       + mu[pos_km1] );
        xmu2     = 2.0/(  mu[pos]       + mu[pos_jm1] );
        xmu3     = 2.0/(  mu[pos]       + mu[pos_ip1] );
        xl       = xl  +  xm;

        qpa      = 0.0625*( qp[pos]     + qp[pos_ip1] + qp[pos_jm1] + qp[pos_ijk]
                          + qp[pos_km1] + qp[pos_ik1] + qp[pos_jk1] + qp[pos_ijk1] );
		//qpa		= 0.0625*(qp[pos] + qp[pos_ip1] + qp[pos_jm1] + qp[pos_ijk]
		//			+ qp[pos_kp1] + qp[pos_ip1+1] + qp[pos_jm1+1] + qp[pos_ijk+1]);
        h        = 0.0625*( qs[pos]     + qs[pos_ip1] + qs[pos_jm1] + qs[pos_ijk]
                          + qs[pos_km1] + qs[pos_ik1] + qs[pos_jk1] + qs[pos_ijk1] );
		//h		= 0.0625*(qs[pos] + qs[pos_ip1] + qs[pos_jm1] + qs[pos_ijk]
		//				+ qs[pos_kp1] + qs[pos_ip1+1] + qs[pos_jm1+1] + qs[pos_ijk+1]);
        h1       = 0.250*(  qs[pos]     + qs[pos_km1] );
		//h1		= 0.250*(qs[pos] + qs[pos_kp1]);
        h2       = 0.250*(  qs[pos]     + qs[pos_jm1] );
        h3       = 0.250*(  qs[pos]     + qs[pos_ip1] );

        h        = -xm*h*d_dh1;
        h1       = -xmu1*h1*d_dh1;
        h2       = -xmu2*h2*d_dh1;
        h3       = -xmu3*h3*d_dh1;
        qpa      = -qpa*xl*d_dh1;
        xm       = xm*d_dth;
        xmu1     = xmu1*d_dth;
        xmu2     = xmu2*d_dth;
        xmu3     = xmu3*d_dth;
        xl       = xl*d_dth;
        f_vx2    = f_vx2*f_vx1;
        h        = h*f_vx1;
        h1       = h1*f_vx1;
        h2       = h2*f_vx1;
        h3       = h3*f_vx1;
        qpa      = qpa*f_vx1;

        xm       = xm+d_DT*h;
        xmu1     = xmu1+d_DT*h1;
        xmu2     = xmu2+d_DT*h2;
        xmu3     = xmu3+d_DT*h3;
        vx1      = d_DT*(1+f_vx2);
        
        u1_ip2   = u1_ip1;
        u1_ip1   = f_u1;
        f_u1     = u1_im1;
        u1_im1   = u1[pos_im1];
        v1_ip1   = f_v1;
        f_v1     = v1_im1;
        v1_im1   = v1_im2;
        v1_im2   = v1[pos_im2];
        w1_ip1   = f_w1;
        f_w1     = w1_im1;
        w1_im1   = w1_im2;
        w1_im2   = w1[pos_im2];

        if(k == d_nzt+align-1)
        {
		u1[pos_kp1] = f_u1 - (f_w1        - w1_im1);
    		v1[pos_kp1] = f_v1 - (w1[pos_jp1] - f_w1);

                g_i  = d_nxt*rankx + i - 4*loop - 1;

                //Interpretation: do not consider gradients over w1 going over domain boundary
                //for u1 and v1, set value that would be outside of boundary to zero
 
                /*original implementation*/
    		/*if(g_i<NX)
        		vs1	= u1_ip1 - (w1_ip1    - f_w1);
    		else
        		vs1	= 0.0;

                g_i  = d_nyt*ranky + j - 4*loop - 1;
    		if(g_i>1)
        		vs2	= v1[pos_jm1] - (f_w1 - w1[pos_jm1]);
    		else
        		vs2	= 0.0;

    		w1[pos_kp1]	= w1[pos_km1] - lam_mu[i*(d_nyt+4+8*loop) + j]*((vs1         - u1[pos_kp1]) + (u1_ip1 - f_u1)
                                      +     			                (v1[pos_kp1] - vs2)         + (f_v1   - v1[pos_jm1]) );*/

//              w1[i,j,k+1] =
//                w1[i,j,k-1] - lambda/(lambda + 2 mu) *
//                u1[i+1,j,k] - u1[i,j,k+1] + v1[i,j,k+1] - u1[i,j,k] - w1[i+1,j,k] + w1[i,j,k] + u1[i+1,j,k]
//		  + w1[i,j,k] - w1[i,j-1,k] + v1(i,j,k)

                /* new implementation */

    		/*if(g_i<NX)
        		vs1	= u1_ip1 - (w1_ip1    - f_w1);
    		else
        		vs1	= 0.0;

                g_i  = d_nyt*ranky + j - 4*loop - 1;
    		if(g_i>1)
        		vs2	= v1[pos_jm1] - (f_w1 - w1[pos_jm1]);
    		else
        		vs2	= 0.0;*/

                float al, amu, a, b, a5, b5;
                register int pos_jm1_kp1, pos_ip1_kp1;
                pos_jm1_kp1 = pos_jm1 + 1;
                pos_ip1_kp1 = pos_ip1 + 1;

                al=1./(0.5*(lam[pos] + lam[pos_km1]));
                amu=1./(0.5*(mu[pos] + mu[pos_km1]));

                a=al;
                b=al+2*amu;

                al=1./(0.5*(lam[pos] + lam[pos_km1]));
                amu=1./(0.5*(mu[pos] + mu[pos_km1]));

                a5=al;
                b5=al+2*amu;

                //may still need workaround for i,j close to boundary
    		w1[pos_kp1] =  w1[pos] - a/b *(u1[pos_ip1_kp1] - u1[pos_kp1] + v1[pos_kp1] - v1[pos_jm1_kp1]) 
                              -(w1[pos] - w1[pos_km1])
                              -a5/b5 * (u1[pos_ip1] - u1[pos] + v1[pos] -v1[pos_jm1]);

        }
	else if(k == d_nzt+align-2)
	{
                u1[pos_kp2] = u1[pos_kp1] - (w1[pos_kp1]   - w1[pos_im1+1]);
                v1[pos_kp2] = v1[pos_kp1] - (w1[pos_jp1+1] - w1[pos_kp1]);
	}
 
    	vs1      = d_c1*(u1_ip1 - f_u1)        + d_c2*(u1_ip2      - u1_im1);
        vs2      = d_c1*(f_v1   - v1[pos_jm1]) + d_c2*(v1[pos_jp1] - v1[pos_jm2]);
        vs3      = d_c1*(f_w1   - w1[pos_km1]) + d_c2*(w1[pos_kp1] - w1[pos_km2]);
 
        tmp      = xl*(vs1+vs2+vs3);
        a1       = qpa*(vs1+vs2+vs3);
        tmp      = tmp+d_DT*a1;

        f_r      = r1[pos];
        xx[pos]  = (xx[pos]  + tmp - xm*(vs2+vs3) + vx1*f_r)*f_dcrj;
        r1[pos]  = f_vx2*f_r - h*(vs2+vs3)        + a1;
        f_r      = r2[pos];
        yy[pos]  = (yy[pos]  + tmp - xm*(vs1+vs3) + vx1*f_r)*f_dcrj;
        r2[pos]  = f_vx2*f_r - h*(vs1+vs3)        + a1;
        f_r      = r3[pos];
        zz[pos]  = (zz[pos]  + tmp - xm*(vs1+vs2) + vx1*f_r)*f_dcrj;
        r3[pos]  = f_vx2*f_r - h*(vs1+vs2)        + a1;

        vs1      = d_c1*(u1[pos_jp1] - f_u1)   + d_c2*(u1[pos_jp2] - u1[pos_jm1]);
        vs2      = d_c1*(f_v1        - v1_im1) + d_c2*(v1_ip1      - v1_im2);
        f_r      = r4[pos];
        xy[pos]  = (xy[pos]  + xmu1*(vs1+vs2) + vx1*f_r)*f_dcrj;
        r4[pos]  = f_vx2*f_r + h1*(vs1+vs2);
  
        if(k == d_nzt+align-1)
        {
                zz[pos+1] = -zz[pos];
        	xz[pos]   = 0.0;
                yz[pos]   = 0.0;
        }
        else
        {
        	vs1     = d_c1*(u1[pos_kp1] - f_u1)   + d_c2*(u1[pos_kp2] - u1[pos_km1]);
        	vs2     = d_c1*(f_w1        - w1_im1) + d_c2*(w1_ip1      - w1_im2);
        	f_r     = r5[pos];
        	xz[pos] = (xz[pos]  + xmu2*(vs1+vs2) + vx1*f_r)*f_dcrj;
        	r5[pos] = f_vx2*f_r + h2*(vs1+vs2);
	 

        	vs1     = d_c1*(v1[pos_kp1] - f_v1) + d_c2*(v1[pos_kp2] - v1[pos_km1]);
        	vs2     = d_c1*(w1[pos_jp1] - f_w1) + d_c2*(w1[pos_jp2] - w1[pos_jm1]);
        	f_r     = r6[pos];
        	yz[pos] = (yz[pos]  + xmu3*(vs1+vs2) + vx1*f_r)*f_dcrj;
        	r6[pos] = f_vx2*f_r + h3*(vs1+vs2);

                if(k == d_nzt+align-2)
                {
                    zz[pos+3] = -zz[pos];
                    xz[pos+2] = -xz[pos];
                    yz[pos+2] = -yz[pos];                                               
		}
		else if(k == d_nzt+align-3)
		{
                    xz[pos+4] = -xz[pos];
                    yz[pos+4] = -yz[pos];
		}
 	}
        pos     = pos_im1;
    }
    return;
}


__global__ void addsrc_cu(int i,      int READ_STEP, int dim,    int npsrc, int* psrc,  int igreen, int nzt,
                          float* d1,  float* u1,     float* v1,  float* w1,
                          float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
                          float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz)
{
        register float vtst, vtst1, tmp;
        register int idx, idy, idz, j, pos, pos_nzt;
        j = blockIdx.x*blockDim.x+threadIdx.x;
        if(j >= npsrc) return;
        vtst = d_DT/(d_DH*d_DH*d_DH);
        vtst1 = 1.0/(d_DH*d_DH);

        i   = i - 1;
        idx = psrc[j*dim]   + 1 + 4*loop;
        idy = psrc[j*dim+1] + 1 + 4*loop;
        idz = psrc[j*dim+2] + align - 1;
        pos = idx*d_slice_1 + idy*d_yline_1 + idz;
        pos_nzt = pos - idz + nzt+align-1;

        if(igreen == -1)
        {
          xx[pos] = xx[pos] - vtst*axx[j*READ_STEP+i];
          yy[pos] = yy[pos] - vtst*ayy[j*READ_STEP+i];
          zz[pos] = zz[pos] - vtst*azz[j*READ_STEP+i];
          xz[pos] = xz[pos] - vtst*axz[j*READ_STEP+i];
          yz[pos] = yz[pos] - vtst*ayz[j*READ_STEP+i];
          xy[pos] = xy[pos] - vtst*axy[j*READ_STEP+i];
        }
        else if(igreen == 1)
        {
          u1[pos] = u1[pos] + vtst*axx[j*READ_STEP+i]/d1[pos];
        }
        else if(igreen == 2)
        {
          v1[pos] = v1[pos] + vtst*ayy[j*READ_STEP+i]/d1[pos];
        }
        else if(igreen == 3)
        {
          w1[pos] = w1[pos] + vtst*azz[j*READ_STEP+i]/d1[pos];
        }
        else if(igreen == -2)
        {
          u1[pos] = u1[pos] + vtst*axx[j*READ_STEP+i]/d1[pos];
          v1[pos] = v1[pos] + vtst*ayy[j*READ_STEP+i]/d1[pos];
          w1[pos] = w1[pos] + vtst*azz[j*READ_STEP+i]/d1[pos];
        }
        else if(igreen == 4){
          tmp = vtst1*axx[j*READ_STEP+i];
          xz[pos_nzt] = tmp;
          tmp = tmp*2;
          xz[pos_nzt+1] = tmp - xz[pos_nzt-1];
          xz[pos_nzt+2] = tmp - xz[pos_nzt-2];
        }
        else if(igreen == 5){
          tmp = vtst1*ayy[j*READ_STEP+i];
          yz[pos_nzt] = tmp;
          tmp = tmp*2;
          yz[pos_nzt+1] = tmp - yz[pos_nzt-1];
          yz[pos_nzt+2] = tmp - yz[pos_nzt-2];
        }
        else if(igreen == 6){
          tmp = 2.0*vtst1*azz[j*READ_STEP+i];
          zz[pos_nzt+1] = tmp - zz[pos_nzt];
          zz[pos_nzt+2] = tmp - zz[pos_nzt-1];
        }


        return;
}
